#include "hip/hip_runtime.h"
#include "cuda_samp1.cuh"

__global__ void addKernel(int n, int* c, const int* a, const int* b)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size, std::string* error_message)
{
  int* dev_a = 0;
  int* dev_b = 0;
  int* dev_c = 0;
  hipError_t cuda_status;

  // Choose which GPU to run on, change this on a multi-GPU system.
  cuda_status = hipSetDevice(0);
  if (cuda_status != hipSuccess) {
    *error_message = "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
    goto Error;
  }

  // Allocate GPU buffers for three vectors (two input, one output)    .
  cuda_status = hipMalloc((void**)&dev_c, size * sizeof(int));
  if (cuda_status != hipSuccess) {
    *error_message = "hipMalloc failed!";
    goto Error;
  }

  cuda_status = hipMalloc((void**)&dev_a, size * sizeof(int));
  if (cuda_status != hipSuccess) {
    *error_message = "hipMalloc failed!";
    goto Error;
  }

  cuda_status = hipMalloc((void**)&dev_b, size * sizeof(int));
  if (cuda_status != hipSuccess) {
    *error_message = "hipMalloc failed!";
    goto Error;
  }

  // Copy input vectors from host memory to GPU buffers.
  cuda_status = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
    *error_message = "hipMemcpy failed!";
    goto Error;
  }

  cuda_status = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
    *error_message = "hipMemcpy failed!";
    goto Error;
  }

  // Launch a kernel on the GPU with one thread for each element.
  addKernel<<<1, 512>>> (size, dev_c, dev_a, dev_b);

  // Check for any errors launching the kernel
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    *error_message = "addKernel launch failed: " + std::string(hipGetErrorString(cuda_status));
    goto Error;
  }

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess) {
    *error_message = "hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
    goto Error;
  }

  // Copy output vector from GPU buffer to host memory.
  cuda_status = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
  if (cuda_status != hipSuccess) {
    *error_message = "hipMemcpy failed!";
    goto Error;
  }

Error:
  hipFree(dev_c);
  hipFree(dev_a);
  hipFree(dev_b);

  return cuda_status;
}
