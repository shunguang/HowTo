#include "hip/hip_runtime.h"
#include "cuda_samp2.cuh"


__global__ void addKernel2(int n, int* c, const int* a, const int* b)
{
#if 0  
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
#else
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    c[i] = a[i] + b[i];    
#endif
}

//assume a,b,c are dev memo address
hipError_t addWithCuda2(int* dev_c, const int* dev_a, const int* dev_b, unsigned int size, std::string* error_message)
{
  hipError_t cuda_status;

  // Launch a kernel on the GPU with one thread for each element.
  int blockSize = 512;
  int numBlocks = (size + blockSize - 1) / blockSize;

  printf("addWithCuda2(): numBlocks=%d, blockSize=%d\n", numBlocks, blockSize);
  addKernel2 << <numBlocks, blockSize >> > (size, dev_c, dev_a, dev_b);
  //addKernel2 << <1, 256 >> > (size, dev_c, dev_a, dev_b);

  // Check for any errors launching the kernel
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    *error_message = "addKernel launch failed: " + std::string(hipGetErrorString(cuda_status));
    goto Error;
  }

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess) {
    *error_message = "hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
    goto Error;
  }

Error:
  return cuda_status;
}


void addWithCpu(int* c, const int* a, const int* b, unsigned int size, std::string* error_message)
{
  for (unsigned int i = 0; i < size; ++i) {
    c[i] = a[i] + b[i];
  }
}

